
#include "../inc/gpu_memory_allocation.h"

void malloc_graph_gpu_memory(Graph &g,G_pointers &p){
    chkerr(hipMalloc(&(p.neighbors),g.neighbors_offset[g.V]*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors,g.neighbors,g.neighbors_offset[g.V]*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.neighbors_offset),(g.V+1)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.neighbors_offset,g.neighbors_offset,(g.V+1)*sizeof(unsigned int),hipMemcpyHostToDevice));
    chkerr(hipMalloc(&(p.degrees),(g.V)*sizeof(unsigned int)));
    chkerr(hipMemcpy(p.degrees,g.degrees,(g.V)*sizeof(unsigned int),hipMemcpyHostToDevice));
    // std::cout<<"memory graph p = "<<p.neighbors[0]<<endl;
}

void get_results_from_gpu(Graph &g,G_pointers &p){
    chkerr(hipMemcpy(g.degrees,p.degrees,(g.V)*sizeof(unsigned int),hipMemcpyDeviceToHost));    
}

void free_graph_gpu_memory(G_pointers &p){
    chkerr(hipFree(p.neighbors));
    chkerr(hipFree(p.neighbors_offset));
    chkerr(hipFree(p.degrees));
}
